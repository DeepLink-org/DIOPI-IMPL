#include "hip/hip_runtime.h"
/**************************************************************************************************
 * Copyright (c) 2022, SenseTime Inc.
 * License
 * Author
 *
 *************************************************************************************************/

#include <diopi/functions.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime.h>

#include "helper.hpp"

#define dispatch_dtype(fun, dtype, gridSize, blockSize, stream, ...)                             \
    if (diopi_dtype_int32 == dtype) {                                                            \
        fun<int32_t><<<gridSize, blockSize, 0, stream>>>(__VA_ARGS__);                           \
    } else if (diopi_dtype_uint32 == dtype) {                                                    \
        fun<uint32_t><<<gridSize, blockSize, 0, stream>>>(__VA_ARGS__);                          \
    } else if (diopi_dtype_int16 == dtype) {                                                      \
        fun<int16_t><<<gridSize, blockSize, 0, stream>>>(__VA_ARGS__);                           \
    } else if (diopi_dtype_uint16 == dtype) {                                                     \
        fun<uint16_t><<<gridSize, blockSize, 0, stream>>>(__VA_ARGS__);                          \
    } else if (diopi_dtype_int8 == dtype) {                                                       \
        fun<int8_t><<<gridSize, blockSize, 0, stream>>>(__VA_ARGS__);                            \
    } else if (diopi_dtype_uint8 == dtype) {                                                      \
        fun<uint8_t><<<gridSize, blockSize, 0, stream>>>(__VA_ARGS__);                           \
    } else if (diopi_dtype_float32 == dtype) {                                                    \
        fun<float><<<gridSize, blockSize, 0, stream>>>(__VA_ARGS__);                             \
    } else if (diopi_dtype_float64 == dtype) {                                                    \
        fun<double><<<gridSize, blockSize, 0, stream>>>(__VA_ARGS__);                            \
    } else if (diopi_dtype_bool == dtype) {                                                       \
        fun<bool><<<gridSize, blockSize, 0, stream>>>(__VA_ARGS__);                              \
    } else {                                                                                     \
        fprintf(stderr, "%s:%s: %s<%s %d><<<%d,%d>>>(%s)", __FILE__, __FUNCTION__, #fun, #dtype, \
                dtype, gridSize, blockSize, #__VA_ARGS__);                                       \
        return diopiDtypeNotSupported;                                                           \
    }

template<typename T> __global__
void vecAdd(const void* a, const void* b, void* c, const int numel, const T alpha) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    const T* A = static_cast<const T*>(a);
    const T* B = static_cast<const T*>(b);
    T* C = static_cast<T*>(c);
    if (id < numel) {
        C[id] = A[id] + alpha * B[id];
    }
}

template<typename T> __global__
void vecAddBroadcast(const void* a, const void* b, void* c, const int numel, const T alpha,
        const int64_t* stride1, const int64_t* stride2, const int64_t* outStride, const int len) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    const T* A = static_cast<const T*>(a);
    const T* B = static_cast<const T*>(b);
    T* C = static_cast<T*>(c);
    int size = id;
    size_t idxA = 0;
    size_t idxB = 0;
    if (id < numel) {
        for (int i = 0; i < len; ++i) {
            int tmp = size / outStride[i];
            idxA += tmp * stride1[i];
            idxB += tmp * stride2[i];
            size = size % outStride[i];
        }
        C[id] = A[idxA] + alpha * B[idxB];
    }
}

template<typename T> __global__
void vecAddScalar(const void* a, const T b, void* c, const int numel, const T alpha) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    const T* A = static_cast<const T*>(a);
    T* C = static_cast<T*>(c);
    if (id < numel) {
        C[id] = A[id] + alpha * b;
    }
}

bool compareShape(const diopiSize_t& size1, const diopiSize_t& size2) {
    if (size1.len == size2.len) {
        for (int i = 0; i < size1.len; ++i) {
            if (size1.data[i] != size2.data[i]) {
                return 0;
            }
        }
        return 1;
    }
    return 0;
}

void computeStride(const diopiSize_t& size1, const diopiSize_t& size2, diopiSize_t outSize,
        int64_t* stride1, int64_t* stride2) {
    int length = size1.len;
    int len = outSize.len;
    int64_t stride = 1;
    for (int i = 0; i < len; ++i) {
        stride1[i] = 0;
        stride2[i] = 0;
    }
    for (int i = 1; i < length + 1; ++i) {
        if (size1.data[length - i] == outSize.data[len - i]) {
            stride1[len - i] = stride;
            stride *= outSize.data[len - i];
        }
    }
    length = size2.len;
    stride = 1;
    for (int i = 1; i < length + 1; ++i) {
        if (size2.data[length - i] == outSize.data[len - i]) {
            stride2[len - i] = stride;
            stride *= outSize.data[len - i];
        }
    }
}

extern "C" diopiError_t diopiAdd(diopiContextHandle_t ctx, diopiTensorHandle_t out,
        diopiConstTensorHandle_t input, diopiConstTensorHandle_t other, const diopiScalar_t* alpha) {
    auto stream  = impl::cuda::getStream(ctx);
    auto trInput = impl::cuda::makeTensor(input);
    auto trOther = impl::cuda::makeTensor(other);
    auto trOut   = impl::cuda::makeTensor(out);

    int blockSize = 256;
    double coff = 0.0;
    if (trInput.dtype() <= 7) {
        coff = alpha->ival;
    } else {
        coff = alpha->fval;
    }
    diopiSize_t inShape = trInput.shape();
    diopiSize_t othShape = trOther.shape();
    int gridSize  = (trOut.numel() + blockSize - 1) / blockSize;
    if (compareShape(inShape, othShape)) {
        dispatch_dtype(vecAdd, trInput.dtype(), gridSize, blockSize, stream,
            trInput.data(), trOther.data(), trOut.data(), trInput.numel(), coff);
    } else {
        diopiSize_t outShape = trOut.shape();
        diopiSize_t outStrideHost = trOut.stride();
        int len = outShape.len;
        int64_t nbytes = len * sizeof(int64_t);

        std::vector<int64_t> inStrideHost(len);
        std::vector<int64_t> othStrideHost(len);
        auto inStride = impl::cuda::requiresBuffer(ctx, nbytes);
        auto othStride = impl::cuda::requiresBuffer(ctx, nbytes);
        auto outStride = impl::cuda::requiresBuffer(ctx, nbytes);

        computeStride(inShape, othShape, outShape, inStrideHost.data(), othStrideHost.data());
        hipMemcpyAsync(inStride.data(), inStrideHost.data(), nbytes, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(othStride.data(), othStrideHost.data(), nbytes, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(outStride.data(), outStrideHost.data, nbytes, hipMemcpyHostToDevice, stream);

        dispatch_dtype(vecAddBroadcast, trInput.dtype(), gridSize, blockSize, stream,
           trInput.data(), trOther.data(), trOut.data(), trOut.numel(), coff, static_cast<const int64_t*>(inStride.data()),
           static_cast<const int64_t*>(othStride.data()), static_cast<const int64_t*>(outStride.data()), len);
    }
    return diopiSuccess;
}

extern "C" diopiError_t diopiAddScalar(diopiContextHandle_t ctx, diopiTensorHandle_t out,
        diopiConstTensorHandle_t input, const diopiScalar_t* other, const diopiScalar_t* alpha) {
    auto stream  = impl::cuda::getStream(ctx);
    auto trInput = impl::cuda::makeTensor(input);
    auto trOut   = impl::cuda::makeTensor(out);
    int blockSize = 256;
    double coff = 0.0;
    double otherVal = 0.0;
    if (trInput.dtype() <= 7) {
        coff = alpha->ival;
        otherVal = other->ival;
    } else {
        coff = alpha->fval;
        otherVal = other->fval;
    }
    int gridSize = (trInput.numel() + blockSize - 1) / blockSize;
    dispatch_dtype(vecAddScalar, trInput.dtype(), gridSize, blockSize, stream,
        trInput.data(), otherVal, trOut.data(), trInput.numel(), coff);
    return diopiSuccess;
}

template<typename T> __global__
void vecFill(void* a, const float value, const int numel) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    T* A = static_cast<T*>(a);
    if (id < numel) {
        A[id] = static_cast<T>(value);
    }
}

extern "C" diopiError_t diopiFill(diopiContextHandle_t ctx, diopiTensorHandle_t input, const diopiScalar_t* value) {
    auto stream = impl::cuda::getStream(ctx);
    auto tr = impl::cuda::makeTensor(input);

    diopiDevice_t device = tr.device();
    diopiDtype_t  dtype  = tr.dtype();
    int64_t       numel  = tr.numel();
    float val;
    if (value->stype <= 7) {
        val = value->ival;
    } else {
        val = value->fval;
    }
    if (diopi_host == device) {
        return diopiErrorOccurred;
    } else {
        int blockSize = 256;
        int gridSize  = (numel + blockSize - 1) / blockSize;
        dispatch_dtype(vecFill, dtype, gridSize, blockSize, stream, tr.data(), val, numel);
    }

    return diopiSuccess;
}


#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

#define CUDA_2D_KERNEL_LOOP(i, n, j, m)                             \
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);   \
       i += blockDim.x * gridDim.x)                                 \
    for (size_t j = blockIdx.y * blockDim.y + threadIdx.y; j < (m); \
         j += blockDim.y * gridDim.y)

#define CUDA_2D_KERNEL_BLOCK_LOOP(i, n, j, m)          \
  for (size_t i = blockIdx.x; i < (n); i += gridDim.x) \
    for (size_t j = blockIdx.y; j < (m); j += gridDim.y)

#define THREADS_PER_BLOCK 512

inline int GET_BLOCKS(const int N, const int num_threads = THREADS_PER_BLOCK) {
  int optimal_block_num = (N + num_threads - 1) / num_threads;
  int max_block_num = 4096;
  return min(optimal_block_num, max_block_num);
}

template <typename T>
__device__ T bilinear_interpolate(const T* input, const int height,
                                  const int width, T y, T x,
                                  const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) return 0;

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  int y_low = (int)y;
  int x_low = (int)x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = input[y_low * width + x_low];
  T v2 = input[y_low * width + x_high];
  T v3 = input[y_high * width + x_low];
  T v4 = input[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height, const int width, T y, T x, T& w1, T& w2, T& w3, T& w4,
    int& x_low, int& x_high, int& y_low, int& y_high,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  y_low = (int)y;
  x_low = (int)x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  // T v1 = input[y_low * width + x_low];
  // T v2 = input[y_low * width + x_high];
  // T v3 = input[y_high * width + x_low];
  // T v4 = input[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

#define MAX_SHARED_SCALAR_T 6144  // 49152 / 8 = 6144

template <typename scalar_t>
__global__ void chamfer_distance_forward_cuda_kernel(int b, int n,
                                                     const scalar_t* xyz, int m,
                                                     const scalar_t* xyz2,
                                                     scalar_t* result,
                                                     int* result_i) {
  __shared__ scalar_t buf[MAX_SHARED_SCALAR_T];
  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int k2 = 0; k2 < m; k2 += THREADS_PER_BLOCK) {
      int end_k = min(m, k2 + THREADS_PER_BLOCK) - k2;
      for (int j = threadIdx.x; j < end_k * 2; j += blockDim.x) {
        buf[j] = xyz2[(i * m + k2) * 2 + j];
      }
      __syncthreads();
      for (int j = threadIdx.x; j < n; j += blockDim.x * gridDim.y) {
        scalar_t x1 = xyz[(i * n + j) * 2 + 0];
        scalar_t y1 = xyz[(i * n + j) * 2 + 1];
        int best_i = 0;
        scalar_t best = 1e10;
        int end_ka = end_k & (~2);
        if (end_ka == THREADS_PER_BLOCK) {
          for (int k = 0; k < THREADS_PER_BLOCK; k += 4) {
#pragma unroll
            for (int j = 0; j < 4; ++j) {
              scalar_t x2 = buf[(k + j) * 2] - x1;
              scalar_t y2 = buf[(k + j) * 2 + 1] - y1;
              scalar_t d = x2 * x2 + y2 * y2;
              if (d < best) {
                best = d;
                best_i = k + k2 + j;
              }
            }
          }
        } else {
          for (int k = 0; k < end_ka; k += 4) {
#pragma unroll
            for (int j = 0; j < 4; ++j) {
              scalar_t x2 = buf[(k + j) * 2] - x1;
              scalar_t y2 = buf[(k + j) * 2 + 1] - y1;
              scalar_t d = x2 * x2 + y2 * y2;
              if (d < best) {
                best = d;
                best_i = k + k2 + j;
              }
            }
          }
        }
        for (int k = end_ka; k < end_k; k++) {
          scalar_t x2 = buf[k * 2 + 0] - x1;
          scalar_t y2 = buf[k * 2 + 1] - y1;
          scalar_t d = x2 * x2 + y2 * y2;
          if (k == 0 || d < best) {
            best = d;
            best_i = k + k2;
          }
        }
        if (k2 == 0 || result[(i * n + j)] > best) {
          result[(i * n + j)] = best;
          result_i[(i * n + j)] = best_i;
        }
      }
      __syncthreads();
    }
  }
}

template <typename scalar_t>
__global__ void chamfer_distance_backward_cuda_kernel(
    int b, int n, const scalar_t* xyz1, int m, const scalar_t* xyz2,
    const scalar_t* grad_dist1, const int* idx1, scalar_t* grad_xyz1,
    scalar_t* grad_xyz2) {
  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int j = threadIdx.x; j < n; j += blockDim.x * gridDim.y) {
      scalar_t x1 = xyz1[(i * n + j) * 2 + 0];
      scalar_t y1 = xyz1[(i * n + j) * 2 + 1];
      int j2 = idx1[i * n + j];
      scalar_t x2 = xyz2[(i * m + j2) * 2 + 0];
      scalar_t y2 = xyz2[(i * m + j2) * 2 + 1];
      scalar_t g = grad_dist1[i * n + j] * 2;
      atomicAdd(&(grad_xyz1[(i * n + j) * 2 + 0]), g * (x1 - x2));
      atomicAdd(&(grad_xyz1[(i * n + j) * 2 + 1]), g * (y1 - y2));
      atomicAdd(&(grad_xyz2[(i * m + j2) * 2 + 0]), -(g * (x1 - x2)));
      atomicAdd(&(grad_xyz2[(i * m + j2) * 2 + 1]), -(g * (y1 - y2)));
    }
  }
}

extern "C" diopiError_t diopiChamferDistance(diopiContextHandle_t ctx, diopiConstTensorHandle_t xyz1_in,
                     diopiConstTensorHandle_t xyz2_in, diopiTensorHandle_t dist1_out,
                     diopiTensorHandle_t dist2_out, diopiTensorHandle_t idx1_out,
                     diopiTensorHandle_t idx2_out) {
  auto xyz1 = impl::cuda::makeTensor(xyz1_in);
  auto xyz2 = impl::cuda::makeTensor(xyz2_in);
  auto dist1 = impl::cuda::makeTensor(dist1_out);
  auto dist2 = impl::cuda::makeTensor(dist2_out);
  auto idx1 = impl::cuda::makeTensor(idx1_out);
  auto idx2 = impl::cuda::makeTensor(idx2_out);
  int batch_size = xyz1.shape(0);
  int n = xyz1.shape(1);
  int m = xyz2.shape(1);
  // here: wait for dipu ready
  // at::cuda::CUDAGuard device_guard(xyz1.device());
  auto stream = impl::cuda::getStream(ctx);
  dispatch_dtype(chamfer_distance_forward_cuda_kernel, xyz1_in.dtype(), GET_BLOCKS(batch_size * n), THREADS_PER_BLOCK, stream,
                batch_size, n, xyz1.data(), m,
                xyz2.data(), dist1.data(),
                static_cast<int*>(idx2.data()));
  dispatch_dtype(chamfer_distance_forward_cuda_kernel, xyz1_in.dtype(), GET_BLOCKS(batch_size * m), THREADS_PER_BLOCK, stream,
                batch_size, m, xyz2.data(), n,
                xyz1.data(), dist2.data(),
                static_cast<int*>(idx2.data()));
  return diopiSuccess;
}

extern "C" diopiError_t diopiChamferDistanceBackward(
    diopiContextHandle_t ctx, diopiConstTensorHandle_t xyz1_in,
    diopiConstTensorHandle_t xyz2_in, diopiConstTensorHandle_t idx1_in,
    diopiConstTensorHandle_t idx2_in, diopiConstTensorHandle_t grad_dist1_in,
    diopiConstTensorHandle_t grad_dist2_in, diopiTensorHandle_t grad_xyz1_out,
    diopiTensorHandle_t grad_xyz2_out) {
  auto xyz1 = impl::cuda::makeTensor(xyz1_in);
  auto xyz2 = impl::cuda::makeTensor(xyz2_in);
  auto idx1 = impl::cuda::makeTensor(idx1_in);
  auto idx2 = impl::cuda::makeTensor(idx2_in);
  auto grad_dist1 = impl::cuda::makeTensor(grad_dist1_in);
  auto grad_dist2 = impl::cuda::makeTensor(grad_dist2);
  auto grad_xyz1 = impl::cuda::makeTensor(grad_xyz1_out);
  auto grad_xyz2 = impl::cuda::makeTensor(grad_xyz2_out);
  int batch_size = xyz1.shape(0);
  int n = xyz1.shape(1);
  int m = xyz2.shape(1);
  // here: wait for dipu ready
  // at::cuda::CUDAGuard device_guard(xyz1.device());
  auto stream = impl::cuda::getStream(ctx);
  dispatch_dtype(chamfer_distance_backward_cuda_kernel, xyz1_in.dtype(), GET_BLOCKS(batch_size * n), THREADS_PER_BLOCK / 2, stream,
                batch_size, m, xyz1.data(), n,
                xyz2.data(), grad_dist1.data(),
                static_cast<int*>(idx1.data()),
                grad_xyz1.data(),
                grad_xyz2.data());
  dispatch_dtype(chamfer_distance_backward_cuda_kernel, xyz1_in.dtype(), GET_BLOCKS(batch_size * m), THREADS_PER_BLOCK / 2, stream,
                batch_size, n, xyz2.data(), m,
                xyz1.data(), grad_dist2.data(),
                static_cast<int*>(idx2.data()),
                grad_xyz2.data(),
                grad_xyz1.data());
  return diopiSuccess;
}