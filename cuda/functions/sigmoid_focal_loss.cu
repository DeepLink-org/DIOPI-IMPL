#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <diopi/functions.h>
#include <diopi/functions_mmcv.h>
#include <stdio.h>
#include<float.h>
#include <iostream>
#include <vector>

#include "../cuda_helper.hpp"
#include "../helper.hpp"

namespace impl {

namespace cuda {


template <typename T>
__global__ void sigmoid_focal_loss_forward_cuda_kernel_diopi(
    const int nthreads, const void* input_, const int64_t* target, const void* weight_,
    void* output_, const T gamma, const T alpha, const int num_classes) {
        const T* input =  static_cast<const T*>(input_);
        T* output = static_cast<T*>(output_);
        const T* weight = static_cast<const T*>(weight_);
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index / num_classes;
    int c = index % num_classes;

    int64_t t = target[n];
    T flag_p = (t == c);
    T flag_n = (t != c);

    // p = sigmoid(x) = 1. / 1. + expf(-x)
    T p = (T)1. / ((T)1. + expf(-input[index]));

    // (1 - p)**gamma * log(p)
    T term_p = pow(((T)1. - p), gamma) * log(max(p, (T)FLT_MIN));
    // p**gamma * log(1 - p)
    T term_n = pow(p, gamma) * log(max((T)1. - p, (T)FLT_MIN));

    output[index] = (T)0.;
    output[index] += -flag_p * alpha * term_p;
    output[index] += -flag_n * ((T)1. - alpha) * term_n;
    if (weight != NULL) {
      output[index] *= weight[t];
    }
  }
}

template <typename T>
__global__ void sigmoid_focal_loss_backward_cuda_kernel(
    const int nthreads, const void* input_, const int64_t* target, const void* weight_,
    void* grad_input_, const T gamma, const T alpha, const int num_classes) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const T* input = static_cast<const T*>(input_);
    T* grad_input = static_cast<T*>(grad_input_);
    const T* weight = static_cast<const T*>(weight_);

    int n = index / num_classes;
    int c = index % num_classes;

    int64_t t = target[n];
    T flag_p = (t == c);
    T flag_n = (t != c);

    // p = sigmoid(x) = 1. / 1. + expf(-x)
    T p = (T)1. / ((T)1. + exp(-input[index]));

    // (1 - p)**gamma * (1 - p - gamma*p*log(p))
    T term_p = pow(((T)1. - p), gamma) *
               ((T)1. - p - (gamma * p * log(max(p, (T)FLT_MIN))));
    // p**gamma * (gamma * (1 - p) * log(1 - p) - p)
    T term_n = pow(p, gamma) *
               (gamma * ((T)1. - p) * log(max((T)1. - p, (T)FLT_MIN)) - p);

    grad_input[index] = (T)0.;
    grad_input[index] += -flag_p * alpha * term_p;
    grad_input[index] += -flag_n * ((T)1. - alpha) * term_n;
    if (weight != NULL) {
      grad_input[index] *= weight[t];
    }
  }
}

}  // namespace cuda

}  // namespace impl

diopiError_t diopiSigmoidFocalLossMmcv(diopiContextHandle_t ctx,
                                          diopiTensorHandle_t input_,
                                          diopiTensorHandle_t target_,
                                          diopiTensorHandle_t weight_,
                                          diopiTensorHandle_t output_,
                                          const float gamma,
                                          const float alpha){
  auto input = impl::cuda::makeTensor(input_);
  auto target = impl::cuda::makeTensor(target_);
  auto weight = impl::cuda::makeTensor(weight_);
  auto output = impl::cuda::makeTensor(output_);

  int output_size = output.numel();
  int num_classes = input.size(1);
  // AT_ASSERTM(target.max().item<int64_t>() <= (int64_t)num_classes,"target label should smaller or equal than num classes");
  auto stream = impl::cuda::getStream(ctx);
  dispatch_float_types_and_half(impl::cuda::sigmoid_focal_loss_forward_cuda_kernel_diopi,
                                input.dtype(),
                                GET_BLOCKS(output_size),
                                THREADS_PER_BLOCK,
                                stream,
                                output_size,
                                input.data(),
                                static_cast<const int64_t*>(target.data()),
                                weight.data(),
                                output.data(),
                                gamma,
                                alpha,
                                num_classes);
  return diopiSuccess;
}

diopiError_t diopiSigmoidFocalLossBackwardMmcv(diopiContextHandle_t ctx,
                                          diopiTensorHandle_t input_,
                                          diopiTensorHandle_t target_,
                                          diopiTensorHandle_t weight_,
                                          diopiTensorHandle_t grad_input_,
                                          const float gamma,
                                          const float alpha){
  auto input = impl::cuda::makeTensor(input_);
  auto target = impl::cuda::makeTensor(target_);
  auto weight = impl::cuda::makeTensor(weight_);
  auto grad_input = impl::cuda::makeTensor(grad_input_);

  int output_size = grad_input.numel();
  int num_classes = input.size(1);
  // AT_ASSERTM(target.max().item<int64_t>() <= (int64_t)num_classes,"target label should smaller or equal than num classes");
  auto stream = impl::cuda::getStream(ctx);
  dispatch_float_types_and_half(impl::cuda::sigmoid_focal_loss_backward_cuda_kernel,
                                input.dtype(),
                                GET_BLOCKS(output_size),
                                THREADS_PER_BLOCK,
                                stream,
                                output_size,
                                input.data(),
                                static_cast<const int64_t*>(target.data()),
                                weight.data(),
                                grad_input.data(),
                                gamma,
                                alpha,
                                num_classes);
  return diopiSuccess;
}