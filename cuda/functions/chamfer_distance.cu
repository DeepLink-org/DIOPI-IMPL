#include "hip/hip_runtime.h"
#include <diopi/functions.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#include "../helper.hpp"
#include "../cuda_helper.hpp"

#define MAX_SHARED_SCALAR_T 6144  // 49152 / 8 = 6144

template <typename scalar_t>
__global__ void chamfer_distance_backward_cuda_kernel_diopi(
    int b, int n, const void* xyz1, int m, const void* xyz2,
    const void* grad_dist1, const int* idx1, void* grad_xyz1,
    void* grad_xyz2) {
  // const scalar_t* xyz1_ = static_cast<const scalar_t*>(xyz1);
  // const scalar_t* xyz2_ = static_cast<const scalar_t*>(xyz2);
  // const scalar_t* grad_dist1_ = static_cast<const scalar_t*>(grad_dist1);
  // scalar_t* grad_xyz1_ = static_cast<scalar_t*>(grad_xyz1);
  // scalar_t* grad_xyz2_ = static_cast<scalar_t*>(grad_xyz2);
  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int j = threadIdx.x; j < n; j += blockDim.x * gridDim.y) {
      const scalar_t* xyz1_ = static_cast<const scalar_t*>(xyz1);
      scalar_t x1 = xyz1_[(i * n + j) * 2 + 0];
      scalar_t y1 = xyz1_[(i * n + j) * 2 + 1];
      int j2 = idx1[i * n + j];
      const scalar_t* xyz2_ = static_cast<const scalar_t*>(xyz2);
      scalar_t x2 = xyz2_[(i * m + j2) * 2 + 0];
      scalar_t y2 = xyz2_[(i * m + j2) * 2 + 1];
      const scalar_t* grad_dist1_ = static_cast<const scalar_t*>(grad_dist1);
      scalar_t g = grad_dist1_[i * n + j] * 2;
      scalar_t* grad_xyz1_ = static_cast<scalar_t*>(grad_xyz1);
      atomicAdd(&(grad_xyz1_[(i * n + j) * 2 + 0]), g * (x1 - x2));
      atomicAdd(&(grad_xyz1_[(i * n + j) * 2 + 1]), g * (y1 - y2));
      scalar_t* grad_xyz2_ = static_cast<scalar_t*>(grad_xyz2);
      atomicAdd(&(grad_xyz2_[(i * m + j2) * 2 + 0]), -(g * (x1 - x2)));
      atomicAdd(&(grad_xyz2_[(i * m + j2) * 2 + 1]), -(g * (y1 - y2)));
    }
  }
}

template <typename scalar_t>
__global__ void chamfer_distance_backward_cuda_kernel_diopi2(
    int b, int n, const void* xyz1_, int m, const void* xyz2_,
    const void* grad_dist1_, const int* idx1, void* grad_xyz1_,
    void* grad_xyz2_) {
  const scalar_t* xyz1 = static_cast<const scalar_t*>(xyz1_);
  const scalar_t* xyz2 = static_cast<const scalar_t*>(xyz2_);
  const scalar_t* grad_dist1 = static_cast<const scalar_t*>(grad_dist1_);
  scalar_t* grad_xyz1 = static_cast<scalar_t*>(grad_xyz1_);
  scalar_t* grad_xyz2 = static_cast<scalar_t*>(grad_xyz2_);

  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int j = threadIdx.x; j < n; j += blockDim.x * gridDim.y) {
      scalar_t x1 = xyz1[(i * n + j) * 2 + 0];
      scalar_t y1 = xyz1[(i * n + j) * 2 + 1];
      int j2 = idx1[i * n + j];
      scalar_t x2 = xyz2[(i * m + j2) * 2 + 0];
      scalar_t y2 = xyz2[(i * m + j2) * 2 + 1];
      scalar_t g = grad_dist1[i * n + j] * 2;
      atomicAdd(&(grad_xyz1[(i * n + j) * 2 + 0]), g * (x1 - x2));
      atomicAdd(&(grad_xyz1[(i * n + j) * 2 + 1]), g * (y1 - y2));
      atomicAdd(&(grad_xyz2[(i * m + j2) * 2 + 0]), -(g * (x1 - x2)));
      atomicAdd(&(grad_xyz2[(i * m + j2) * 2 + 1]), -(g * (y1 - y2)));
    }
  }
}

// DIOPI_API diopiError_t diopiChamferDistance(diopiContextHandle_t ctx, diopiConstTensorHandle_t xyz1_in, diopiConstTensorHandle_t xyz2_in, diopiTensorHandle_t dist1_out,
//                                             diopiTensorHandle_t dist2_out, diopiTensorHandle_t idx1_out, diopiTensorHandle_t idx2_out);

extern "C" diopiError_t diopiChamferDistance(diopiContextHandle_t ctx, diopiConstTensorHandle_t xyz1_in,
                     diopiConstTensorHandle_t xyz2_in, diopiTensorHandle_t dist1_out,
                     diopiTensorHandle_t dist2_out, diopiTensorHandle_t idx1_out,
                     diopiTensorHandle_t idx2_out) {
  auto xyz1 = impl::cuda::makeTensor(xyz1_in);
  auto xyz2 = impl::cuda::makeTensor(xyz2_in);
  auto dist1 = impl::cuda::makeTensor(dist1_out);
  auto dist2 = impl::cuda::makeTensor(dist2_out);
  auto idx1 = impl::cuda::makeTensor(idx1_out);
  auto idx2 = impl::cuda::makeTensor(idx2_out);
  int batch_size = xyz1.size(0);
  std::cout << "dkx fwd batch_size" << batch_size << std::endl;
  int n = xyz1.size(1);
  int m = xyz2.size(1);
  std::cout << "dkx fwd n" << n << std::endl;
  std::cout << "dkx fwd m" << m << std::endl;
  // here: wait for dipu ready
  // // at::cuda::CUDAGuard device_guard(xyz1.device());
  auto stream = impl::cuda::getStream(ctx);
  dispatch_float_types_and_half(chamfer_distance_forward_cuda_kernel_diopi, xyz1.dtype(), GET_BLOCKS(batch_size * n), THREADS_PER_BLOCK, stream,
                batch_size, n, xyz1.data(), m,
                xyz2.data(), dist1.data(),
                static_cast<int*>(idx1.data()));
  dispatch_float_types_and_half(chamfer_distance_forward_cuda_kernel_diopi, xyz1.dtype(), GET_BLOCKS(batch_size * m), THREADS_PER_BLOCK, stream,
                batch_size, m, xyz2.data(), n,
                xyz1.data(), dist2.data(),
                static_cast<int*>(idx2.data()));
  return diopiSuccess;
}

// extern "C" {
//     c10::DeviceType device2DeviceType(const diopiDevice_t device);
// }

// DIOPI_API diopiError_t diopiChamferDistanceBackward(diopiContextHandle_t ctx, diopiConstTensorHandle_t xyz1, diopiConstTensorHandle_t xyz2,
//                                             diopiConstTensorHandle_t idx1, diopiConstTensorHandle_t idx2, diopiConstTensorHandle_t grad_dist1, diopiConstTensorHandle_t grad_dist2,
//                                             diopiTensorHandle_t grad_xyz1, diopiTensorHandle_t grad_xyz2);

// extern c 和声明不一致。 DIOPI_API 这是一个 __attribute__((weak)) 的声明。
extern "C" diopiError_t diopiChamferDistanceBackward(
    diopiContextHandle_t ctx, diopiConstTensorHandle_t xyz1_in,
    diopiConstTensorHandle_t xyz2_in, diopiConstTensorHandle_t idx1_in,
    diopiConstTensorHandle_t idx2_in, diopiConstTensorHandle_t grad_dist1_in,
    diopiConstTensorHandle_t grad_dist2_in, diopiTensorHandle_t grad_xyz1_out,
    diopiTensorHandle_t grad_xyz2_out) {
  auto xyz1 = impl::cuda::makeTensor(xyz1_in);
  auto xyz2 = impl::cuda::makeTensor(xyz2_in);
  auto idx1 = impl::cuda::makeTensor(idx1_in);
  auto idx2 = impl::cuda::makeTensor(idx2_in);
  auto grad_dist1 = impl::cuda::makeTensor(grad_dist1_in);
  auto grad_dist2 = impl::cuda::makeTensor(grad_dist2_in);
  auto grad_xyz1 = impl::cuda::makeTensor(grad_xyz1_out);
  auto grad_xyz2 = impl::cuda::makeTensor(grad_xyz2_out);
  int batch_size = xyz1.size(0);
  std::cout << "dkx bwd batch_size" << batch_size << std::endl;
  int n = xyz1.size(1);
  int m = xyz2.size(1);
  std::cout << "dkx bwd n" << n << std::endl;
  std::cout << "dkx bwd m" << m << std::endl;
  // here: wait for dipu ready
  //// at::cuda::CUDAGuard device_guard(device2DeviceType(xyz1.device()));
  auto stream = impl::cuda::getStream(ctx);
  // dispatch_float_types_and_half(
  //               chamfer_distance_backward_cuda_kernel_diopi,
  //               xyz1.dtype(),
  //               GET_BLOCKS(batch_size * n),
  //               THREADS_PER_BLOCK / 2,
  //               stream,
  //               batch_size, m, xyz1.data(), n,
  //               xyz2.data(), grad_dist1.data(),
  //               static_cast<const int*>(idx1.data()),
  //               grad_xyz1.data(),
  //               grad_xyz2.data());
  dispatch_float_types_and_half(
                chamfer_distance_backward_cuda_kernel_diopi2,
                xyz1.dtype(),
                GET_BLOCKS(batch_size * n), THREADS_PER_BLOCK / 2, stream,
                batch_size, m, xyz1.data(), n,
                xyz2.data(), grad_dist1.data(),
                static_cast<const int*>(idx1.data()), grad_xyz1.data(),
                grad_xyz2.data());
  // dispatch_float_types_and_half(chamfer_distance_backward_cuda_kernel_diopi,
  //               xyz1.dtype(),
  //               GET_BLOCKS(batch_size * m),
  //               THREADS_PER_BLOCK / 2,
  //               stream,
  //               batch_size, n, xyz2.data(), m,
  //               xyz1.data(), grad_dist2.data(),
  //               static_cast<const int*>(idx2.data()),
  //               grad_xyz2.data(),
  //               grad_xyz1.data());
  dispatch_float_types_and_half(chamfer_distance_backward_cuda_kernel_diopi2,
                xyz1.dtype(),
                GET_BLOCKS(batch_size * m), THREADS_PER_BLOCK / 2, stream,
                batch_size, n, xyz2.data(), m,
                xyz1.data(), grad_dist2.data(),
                static_cast<const int*>(idx2.data()), grad_xyz2.data(),
                grad_xyz1.data());
  return diopiSuccess;
}