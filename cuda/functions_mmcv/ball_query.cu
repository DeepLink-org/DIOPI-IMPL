#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <diopi/functions.h>
#include <diopi/functions_mmcv.h>
#include <stdio.h>
#include<assert.h>

#include <iostream>
#include <vector>

#include "../cuda_helper.hpp"
#include "../helper.hpp"

namespace impl {

namespace cuda {

template <typename scalar_t>
__global__ void ball_query_forward_cuda_kernel_diopi(int b, int n, int m,
                                                    float min_radius,
                                                    float max_radius, int nsample,
                                                    const void* new_xyz_, const void* xyz_,
                                                    int* idx) {
  // new_xyz: (B, M, 3)
  // xyz: (B, N, 3)
  // output:
  //      idx: (B, M, nsample)

  const scalar_t* new_xyz=static_cast<const scalar_t*>(new_xyz_);
  const scalar_t* xyz=static_cast<const scalar_t*>(xyz_);

  int bs_idx = blockIdx.y;
  CUDA_1D_KERNEL_LOOP(pt_idx, m) {
    if (bs_idx >= b) return;

    new_xyz += bs_idx * m * 3 + pt_idx * 3;
    xyz += bs_idx * n * 3;
    idx += bs_idx * m * nsample + pt_idx * nsample;

    float max_radius2 = max_radius * max_radius;
    float min_radius2 = min_radius * min_radius;
    scalar_t new_x = new_xyz[0];
    scalar_t new_y = new_xyz[1];
    scalar_t new_z = new_xyz[2];

    int cnt = 0;
    for (int k = 0; k < n; ++k) {
      scalar_t x = xyz[k * 3 + 0];
      scalar_t y = xyz[k * 3 + 1];
      scalar_t z = xyz[k * 3 + 2];
      scalar_t d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
             (new_z - z) * (new_z - z);
      if (d2 == 0 || (d2 >= min_radius2 && d2 < max_radius2)) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[l] = k;
          }
        }
        idx[cnt] = k;
        ++cnt;
        if (cnt >= nsample) break;
      }
    }
  }
}

}  // namespace cuda

}  // namespace impl

DIOPI_API diopiError_t diopiBallQuery(diopiContextHandle_t ctx, diopiTensorHandle_t idx_,
                        diopiConstTensorHandle_t new_xyz_, diopiConstTensorHandle_t xyz_, 
                        int64_t b, int64_t n, int64_t m, int64_t nsample,
                        float min_radius, float max_radius) {
  // new_xyz: (B, M, 3)
  // xyz: (B, N, 3)
  // output:
  //      idx: (B, M, nsample)
  auto new_xyz = impl::cuda::makeTensor(new_xyz_);
  auto xyz = impl::cuda::makeTensor(xyz_);
  auto idx = impl::cuda::makeTensor(idx_);

//   at::cuda::CUDAGuard device_guard(new_xyz.device());
  auto stream = impl::cuda::getStream(ctx);

  // blockIdx.x(col), blockIdx.y(row)
  dim3 blocks(GET_BLOCKS(m, THREADS_PER_BLOCK), b);
  dim3 threads(THREADS_PER_BLOCK);


  dispatch_float_types_and_half(impl::cuda::ball_query_forward_cuda_kernel_diopi,
                                new_xyz.dtype(),
                                blocks,
                                threads,
                                stream,
                                b, 
                                n,
                                m,
                                min_radius, 
                                max_radius, 
                                nsample,
                                new_xyz.data(), 
                                xyz.data(),
                                (int*)(idx.data()));
  return diopiSuccess;
}
