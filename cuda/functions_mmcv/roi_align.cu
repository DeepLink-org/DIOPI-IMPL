#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <diopi/functions.h>
#include <diopi/functions_mmcv.h>
#include <stdio.h>
#include<assert.h>
#include <float.h>

#include <iostream>
#include <vector>

#include "../cuda_helper.hpp"
#include "../helper.hpp"

namespace impl {

namespace cuda {

/*** Forward ***/
template <typename T>
__global__ void roi_align_forward_cuda_kernel_diopi(
    const int nthreads, const void* input_, const void* rois_, void* output_, void* argmax_y_,
    void* argmax_x_, const int pooled_height, const int pooled_width,
    const T spatial_scale, const int sampling_ratio,
    const int pool_mode,  // 0 - max pool, 1 - avg pool
    const bool aligned, const int channels, const int height, const int width) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {

    const T* input = static_cast<const T*>(input_);
    const T* rois = static_cast<const T*>(rois_);
    T* output = static_cast<T*>(output_);
    T* argmax_y = static_cast<T*>(argmax_y_);
    T* argmax_x = static_cast<T*>(argmax_x_);
    


    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_rois = rois + n * 5;
    int roi_batch_ind = offset_rois[0];

    // Do not using rounding; this implementation detail is critical
    T offset = aligned ? (T)0.5 : (T)0.0;
    T roi_start_w = offset_rois[1] * spatial_scale - offset;
    T roi_start_h = offset_rois[2] * spatial_scale - offset;
    T roi_end_w = offset_rois[3] * spatial_scale - offset;
    T roi_end_h = offset_rois[4] * spatial_scale - offset;

    T roi_width = roi_end_w - roi_start_w;
    T roi_height = roi_end_h - roi_start_h;
    if (!aligned) {  // for backward-compatibility only
      roi_width = max(roi_width, (T)1.);
      roi_height = max(roi_height, (T)1.);
    }

    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const T* offset_input =
        input + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h =
        (sampling_ratio > 0)
            ? sampling_ratio
            : static_cast<int>(ceilf(roi_height / pooled_height));
    int roi_bin_grid_w =
        (sampling_ratio > 0)
            ? sampling_ratio
            : static_cast<int>(ceilf(roi_width / pooled_width));

    if (pool_mode == 0) {
      // We do max pooling inside a bin
      T maxval = -FLT_MAX;
      T maxidx_y = -1.f, maxidx_x = -1.f;
      for (int iy = 0; iy < roi_bin_grid_h; iy++) {
        const T y = roi_start_h + ph * bin_size_h +
                    static_cast<T>(iy + .5f) * bin_size_h /
                        static_cast<T>(roi_bin_grid_h);
        for (int ix = 0; ix < roi_bin_grid_w; ix++) {
          const T x = roi_start_w + pw * bin_size_w +
                      static_cast<T>(ix + .5f) * bin_size_w /
                          static_cast<T>(roi_bin_grid_w);
          T val =
              bilinear_interpolate(offset_input, height, width, y, x, index);
          if (val > maxval) {
            maxval = val;
            maxidx_y = y;
            maxidx_x = x;
          }
        }
      }
      output[index] = maxval;
      argmax_y[index] = maxidx_y;
      argmax_x[index] = maxidx_x;
    } else if (pool_mode == 1) {
      // We do average pooling inside a bin
      const T count = max(roi_bin_grid_h * roi_bin_grid_w, 1);
      T output_val = 0.;
      for (int iy = 0; iy < roi_bin_grid_h; iy++) {
        const T y = roi_start_h + ph * bin_size_h +
                    static_cast<T>(iy + .5f) * bin_size_h /
                        static_cast<T>(roi_bin_grid_h);
        for (int ix = 0; ix < roi_bin_grid_w; ix++) {
          const T x = roi_start_w + pw * bin_size_w +
                      static_cast<T>(ix + .5f) * bin_size_w /
                          static_cast<T>(roi_bin_grid_w);
          T val =
              bilinear_interpolate(offset_input, height, width, y, x, index);
          output_val += val;
        }
      }
      output[index] = output_val / count;
    }
  }
}

/*** Backward ***/
template <typename T>
__global__ void roi_align_backward_cuda_kernel_diopi(
    const int nthreads, const void* grad_output_, const void* rois_, const void* argmax_y_,
    const void* argmax_x_, void* grad_input_, const int pooled_height,
    const int pooled_width, const T spatial_scale, const int sampling_ratio,
    const int pool_mode,  // 0 - max pool, 1 - avg pool
    const bool aligned, const int channels, const int height, const int width) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {

    const T* grad_output = static_cast<const T*>(grad_output_);
    const T* rois = static_cast<const T*>(rois_);
    const T* argmax_y = static_cast<const T*>(argmax_y_);
    const T* argmax_x = static_cast<const T*>(argmax_x_);
    T* grad_input = static_cast<T*>(grad_input_);
    
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T grad_output_this_bin = grad_output[index];

    const T* offset_rois = rois + n * 5;
    int roi_batch_ind = offset_rois[0];
    T* offset_grad_input =
        grad_input + ((roi_batch_ind * channels + c) * height * width);

    if (pool_mode == 0) {
      T y = argmax_y[index], x = argmax_x[index];
      if (y != -1.f) {
        T w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;
        bilinear_interpolate_gradient(height, width, y, x, w1, w2, w3, w4,
                                      x_low, x_high, y_low, y_high, index);

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          atomicAdd(offset_grad_input + y_low * width + x_low,
                    grad_output_this_bin * w1);
          atomicAdd(offset_grad_input + y_low * width + x_high,
                    grad_output_this_bin * w2);
          atomicAdd(offset_grad_input + y_high * width + x_low,
                    grad_output_this_bin * w3);
          atomicAdd(offset_grad_input + y_high * width + x_high,
                    grad_output_this_bin * w4);
        }
      }
    } else if (pool_mode == 1) {
      // Do not using rounding; this implementation detail is critical
      T offset = aligned ? (T)0.5 : (T)0.0;
      T roi_start_w = offset_rois[1] * spatial_scale - offset;
      T roi_start_h = offset_rois[2] * spatial_scale - offset;
      T roi_end_w = offset_rois[3] * spatial_scale - offset;
      T roi_end_h = offset_rois[4] * spatial_scale - offset;

      T roi_width = roi_end_w - roi_start_w;
      T roi_height = roi_end_h - roi_start_h;
      if (!aligned) {  // for backward-compatibility only
        roi_width = max(roi_width, (T)1.);
        roi_height = max(roi_height, (T)1.);
      }

      T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
      T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

      // We use roi_bin_grid to sample the grid and mimic integral
      int roi_bin_grid_h =
          (sampling_ratio > 0)
              ? sampling_ratio
              : static_cast<int>(ceilf(roi_height / pooled_height));
      int roi_bin_grid_w =
          (sampling_ratio > 0)
              ? sampling_ratio
              : static_cast<int>(ceilf(roi_width / pooled_width));

      // We do average (integral) pooling inside a bin
      const T count = roi_bin_grid_h * roi_bin_grid_w;  // e.g. = 4

      for (int iy = 0; iy < roi_bin_grid_h; iy++) {
        const T y = roi_start_h + ph * bin_size_h +
                    static_cast<T>(iy + .5f) * bin_size_h /
                        static_cast<T>(roi_bin_grid_h);
        for (int ix = 0; ix < roi_bin_grid_w; ix++) {
          const T x = roi_start_w + pw * bin_size_w +
                      static_cast<T>(ix + .5f) * bin_size_w /
                          static_cast<T>(roi_bin_grid_w);

          T w1, w2, w3, w4;
          int x_low, x_high, y_low, y_high;
          bilinear_interpolate_gradient(height, width, y, x, w1, w2, w3, w4,
                                        x_low, x_high, y_low, y_high, index);

          if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
            atomicAdd(offset_grad_input + y_low * width + x_low,
                      grad_output_this_bin * w1 / count);
            atomicAdd(offset_grad_input + y_low * width + x_high,
                      grad_output_this_bin * w2 / count);
            atomicAdd(offset_grad_input + y_high * width + x_low,
                      grad_output_this_bin * w3 / count);
            atomicAdd(offset_grad_input + y_high * width + x_high,
                      grad_output_this_bin * w4 / count);
          }
        }
      }
    }
  }
}

} // namespace cuda

} // namespace impl


DIOPI_API diopiError_t diopiRoiAlignMmcv(diopiContextHandle_t ctx, diopiTensorHandle_t output_,
                                        diopiTensorHandle_t argmax_y_, diopiTensorHandle_t argmax_x_,diopiTensorHandle_t input_,
                                        diopiTensorHandle_t rois_, int64_t aligned_height, int64_t aligned_width, int64_t sampling_ratio,
                                        int64_t pool_mode, float spatial_scale,  bool aligned){
    auto input=impl::cuda::makeTensor(input_);
    auto rois=impl::cuda::makeTensor(rois_);
    auto output=impl::cuda::makeTensor(output_);
    auto argmax_y=impl::cuda::makeTensor(argmax_y_);
    auto argmax_x=impl::cuda::makeTensor(argmax_x_);

    int output_size = output.numel();
    int channels = input.size(1);
    int height = input.size(2);
    int width = input.size(3);

    //   at::cuda::CUDAGuard device_guard(new_xyz.device());
    auto stream = impl::cuda::getStream(ctx);
    dispatch_float_types_and_half(impl::cuda::roi_align_forward_cuda_kernel_diopi,
                                  input.dtype(),
                                  GET_BLOCKS(output_size),
                                  THREADS_PER_BLOCK,
                                  stream,
                                  output_size,
                                  input.data(),
                                  rois.data(),
                                  output.data(),
                                  argmax_y.data(),
                                  argmax_x.data(),
                                  aligned_height,
                                  aligned_width,
                                  spatial_scale,
                                  sampling_ratio,
                                  pool_mode,
                                  aligned,
                                  channels,
                                  height,
                                  width);
  return diopiSuccess;
}



DIOPI_API diopiError_t diopiRoiAlignBackwardMmcv(diopiContextHandle_t ctx, diopiTensorHandle_t grad_input, diopiTensorHandle_t grad_output,   
                                                diopiTensorHandle_t rois, diopiTensorHandle_t argmax_y,
                                                diopiTensorHandle_t argmax_x, int64_t aligned_height, int64_t aligned_width,
                                                int64_t sampling_ratio, int64_t pool_mode, float spatial_scale, bool aligned){
  auto grad_output_=impl::cuda::makeTensor(grad_output);
  auto rois_=impl::cuda::makeTensor(rois);
  auto argmax_y_=impl::cuda::makeTensor(argmax_y);
  auto argmax_x_=impl::cuda::makeTensor(argmax_x);
  auto grad_input_=impl::cuda::makeTensor(grad_input);

  int output_size = grad_output_.numel();
  int channels = grad_input_.size(1);
  int height = grad_input_.size(2);
  int width = grad_input_.size(3);

  //   at::cuda::CUDAGuard device_guard(new_xyz.device());
  auto stream = impl::cuda::getStream(ctx); 
  dispatch_float_types_and_half(impl::cuda::roi_align_backward_cuda_kernel_diopi,
                                grad_output_.dtype(),
                                GET_BLOCKS(output_size),
                                THREADS_PER_BLOCK,
                                stream,
                                output_size,
                                grad_output_.data(),
                                rois_.data(),
                                argmax_y_.data(),
                                argmax_x_.data(),
                                grad_input_.data(),
                                aligned_height,
                                aligned_width,
                                spatial_scale,
                                sampling_ratio,
                                pool_mode,
                                aligned,
                                channels,
                                height,
                                width);
  return diopiSuccess;
}
